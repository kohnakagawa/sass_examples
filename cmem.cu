#include "hip/hip_runtime.h"
#include <iostream>
#include <algorithm>
#include <hip/hip_runtime_api.h>

__global__ void kernel1(double* a,
                        const double cf) {
  const int tid = threadIdx.x + blockIdx.x * blockDim.x;
  a[tid] *= cf;
}

__constant__ double cf_c;

__global__ void kernel2(double* b) {
  const int tid = threadIdx.x + blockIdx.x * blockDim.x;
  b[tid] *= cf_c;
}

__device__ __noinline__ double add_and_mul(const double a,
                                           const double b,
                                           const double c) {
  const double ret = (a + b) * c;
  return ret;
}

__global__ void kernel3(double* a) {
  const int tid = threadIdx.x + blockIdx.x * blockDim.x;
  const auto ret = add_and_mul(a[tid], a[tid], a[tid]);
  a[tid] = add_and_mul(ret, ret, ret);
}

__global__ void kernel4(double* a) {
  const int tid = threadIdx.x + blockIdx.x * blockDim.x;
  if (tid < 100) {
    a[tid] = 10;
  } else {
    a[tid] = 0;
  }
}

__global__ void kernel5(int64_t* a) {
  const int tid = threadIdx.x + blockIdx.x * blockDim.x;
  a[tid] += a[tid];
}

auto main(const int argc,
          const char* argv[]) -> int {
  const auto tb_size = 128;
  const auto gr_size = 1000;
  const auto ar_size = tb_size * gr_size;

  // allocate
  double* a_h = new double [ar_size];
  double* a_d = nullptr;
  checkCudaErrors(hipMalloc((void**)&a_d,
                             ar_size * sizeof(double)));

  // init
  std::fill(a_h, a_h + ar_size, 2.0);

  const double cf = 2.0;
  checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(cf_c), &cf, sizeof(double)));

  // host -> device
  checkCudaErrors(hipMemcpy(a_d, a_h, ar_size * sizeof(double),
                             hipMemcpyHostToDevice));

  kernel1<<<gr_size, tb_size>>>(a_d, double(argc));
  checkCudaErrors(hipDeviceSynchronize());

  kernel1<<<gr_size, tb_size>>>(a_d, 2.0);
  checkCudaErrors(hipDeviceSynchronize());

  kernel2<<<gr_size, tb_size>>>(a_d);
  checkCudaErrors(hipDeviceSynchronize());

  // device -> host
  checkCudaErrors(hipMemcpy(a_h, a_d, ar_size * sizeof(double),
                             hipMemcpyDeviceToHost));

  // free
  checkCudaErrors(hipFree(a_d));
  delete [] a_h;
}
